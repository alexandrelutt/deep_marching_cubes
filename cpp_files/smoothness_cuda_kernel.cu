#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

/**
 * calculate the loss between two neigboring occupancy status 
 */	
__global__ void occupancy_connectivity_kernel( const float *occupancy, float *loss ){

  int i=blockIdx.x;
  int j=blockIdx.y;
  int k=blockIdx.z;

  int W=gridDim.x-1;
  int H=gridDim.y-1;
  int D=gridDim.z-1;

  float loss_ = 0.0;

  float p1 = occupancy[ i*(H+1)*(D+1) + j*(D+1) + k ]; 
 
  if (j<H){
      float p2 = occupancy[ i*(H+1)*(D+1) + (j+1)*(D+1) + k ]; 
      // l1 loss
      loss_ += abs(p1-p2);
  }
  if (i<W){
      float p3 = occupancy[ (i+1)*(H+1)*(D+1) + j*(D+1) + k ]; 
      // l1 loss
      loss_ += abs(p1-p3);
  }
  if (k<D){
      float p4 = occupancy[ i*(H+1)*(D+1) + j*(D+1) + k+1 ]; 
      // l1 loss
      loss_ += abs(p1-p4);
  }
  loss[ i*(H+1)*(D+1) + j*(D+1) + k ] = loss_;
}

/**
 * propagate the gradient to the occupancy status 
 */	
__global__ void grad_occupancy_connectivity_kernel( const float *occupancy, float *grad_occupancy ){

  int i=blockIdx.x;
  int j=blockIdx.y;
  int k=blockIdx.z;

  int W=gridDim.x-1;
  int H=gridDim.y-1;
  int D=gridDim.z-1;

  float p1 = occupancy[ i*(H+1)*(D+1) + j*(D+1) + k ]; 
 
  if (j<H){
      float p2 = occupancy[ i*(H+1)*(D+1) + (j+1)*(D+1) + k ]; 
      // l1 loss
      float sign;
      if (p1>p2){ sign = 1.0; }else{ sign = -1.0; }
      atomicAdd( &grad_occupancy[ i*(H+1)*(D+1) + j*(D+1) + k ], sign );
      atomicAdd( &grad_occupancy[ i*(H+1)*(D+1) + (j+1)*(D+1) + k ], -sign );

  }
  if (i<W){
      float p3 = occupancy[ (i+1)*(H+1)*(D+1) + j*(D+1) + k ]; 
      // l1 loss
      float sign;
      if (p1>p3){ sign = 1.0; }else{ sign = -1.0; }
      atomicAdd( &grad_occupancy[ i*(H+1)*(D+1) + j*(D+1) + k ], sign );
      atomicAdd( &grad_occupancy[ (i+1)*(H+1)*(D+1) + j*(D+1) + k ], -sign );
  }
  if (k<D){
      float p4 = occupancy[ i*(H+1)*(D+1) + j*(D+1) + k+1 ]; 
      float sign;
      if (p1>p4){ sign = 1.0; }else{ sign = -1.0; }
      atomicAdd( &grad_occupancy[ i*(H+1)*(D+1) + j*(D+1) + k ], sign );
      atomicAdd( &grad_occupancy[ i*(H+1)*(D+1) + j*(D+1) + k+1 ], -sign );
  }
}

void connectivity_cuda_forward(
    torch::Tensor occupancy,
    torch::Tensor loss){

    int N = occupancy.size(0);

    dim3 dimGrid(N, N, N);
    const int threads = 1024;
    
    auto loss_all = torch::empty({N * N * N});
    torch::zeros_like(loss_all);
    
    occupancy_connectivity_kernel<<<dimGrid, threads>>>(
        occupancy.data_ptr<float>(),
        loss_all.data_ptr<float>());

    std::cout << "loss_all: " << loss_all << std::endl;
    torch::Tensor sum_loss = torch::sum(loss_all);
    std::cout << "sum_loss: " << sum_loss << std::endl;
    auto loss_ = sum_loss.item<float>();
    std::cout << "loss_: " << loss_ << std::endl;
    loss[0] = loss_;    
}


void connectivity_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor occupancy,
    torch::Tensor grad_occupancy){

    int N = occupancy.size(0);

    dim3 dimGrid(N, N, N);
    const int threads = 1024;

    grad_occupancy_connectivity_kernel<<<dimGrid, threads>>>(
        occupancy.data_ptr<float>(),
        grad_occupancy.data_ptr<float>());

    float grad_output_ = grad_output[0].item<float>();

    grad_occupancy *= grad_output_;

}